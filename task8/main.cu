#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void HelloWorld()
{
    printf("hello, %d, %d\n", blockIdx.x, threadIdx.x);


}

int main()
{
    HelloWorld <<<2, 5>>>();

    hipDeviceSynchronize();
    return 0;
}