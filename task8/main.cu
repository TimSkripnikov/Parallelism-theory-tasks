#include "hip/hip_runtime.h"
// main.cu

#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <fstream>
#include <memory>
#include <cassert>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <boost/program_options.hpp>

namespace bpo = boost::program_options;

__global__ void jacobi_kernel(double* out, const double* in, int size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < size - 1 && j > 0 && j < size - 1) {
        int idx = i * size + j;
        out[idx] = 0.25 * (
            in[idx + 1] + in[idx - 1] +
            in[idx + size] + in[idx - size]
        );
    }
}

__global__ void compute_diff_kernel(double* diff, const double* A, const double* B, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        diff[idx] = fabs(A[idx] - B[idx]);
    }
}

double lin_interp(double x, double x1, double y1, double x2, double y2) {
    return y1 + ((x - x1) * (y2 - y1)) / (x2 - x1);
}

void init(std::vector<double>& A, int size) {
    A[0] = 10.0;
    A[size - 1] = 20.0;
    A[(size - 1) * size + (size - 1)] = 30.0;
    A[(size - 1) * size] = 20.0;

    for (int i = 1; i < size - 1; ++i) {
        A[i] = lin_interp(i, 0.0, A[0], size - 1, A[size - 1]);
        A[i * size] = lin_interp(i, 0.0, A[0], size - 1, A[(size - 1) * size]);
        A[i * size + (size - 1)] = lin_interp(i, 0.0, A[size - 1], size - 1, A[(size - 1) * size + (size - 1)]);
        A[(size - 1) * size + i] = lin_interp(i, 0.0, A[(size - 1) * size], size - 1, A[(size - 1) * size + (size - 1)]);
    }
}

double compute_max_error(double* A, double* B, int N) {
    double* d_diff;

    hipMalloc(&d_diff, N * sizeof(double));

    int threads = 256;

    int blocks = (N + threads - 1) / threads;

    compute_diff_kernel<<<blocks, threads>>>(d_diff, A, B, N);


    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;


    double* d_result;
    hipMalloc(&d_result, sizeof(double));

    hipcub::DeviceReduce::Max(
        d_temp_storage, 
        temp_storage_bytes, 
        d_diff, 
        d_result, 
        N
    );

    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    hipcub::DeviceReduce::Max(
        d_temp_storage, 
        temp_storage_bytes, 
        d_diff, 
        d_result, 
        N
    );

    double h_result;

    hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_diff);
    hipFree(d_temp_storage);
    hipFree(d_result);
    return h_result;
}

void save_to_file(const std::vector<double>& A, int size, const std::string& filename) {
    std::ofstream f(filename);
    if (!f.is_open()) {
        std::cerr << "Failed to open file for writing: " << filename << std::endl;
        return;
    }

    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            f << A[i * size + j] << " ";
        }
        f << "\n";
    }
    f.close();
}

int main(int argc, char** argv) {
    int size, max_iters;
    double eps;

    bpo::options_description desc("Options");
    desc.add_options()
        ("help,h", "Show help")
        ("size", bpo::value<int>(&size)->default_value(128), "Grid size")
        ("num_iters", bpo::value<int>(&max_iters)->default_value(1000000), "Max iterations")
        ("eps", bpo::value<double>(&eps)->default_value(1e-6), "Precision");

    bpo::variables_map vm;
    bpo::store(bpo::parse_command_line(argc, argv, desc), vm);
    bpo::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << std::endl;
        return 0;
    }

    int N = size * size;

    std::vector<double> host_A(N, 0.0), host_B(N, 0.0);
    init(host_A, size);
    init(host_B, size);

    double *d_A, *d_B;

    hipMalloc(&d_A, N * sizeof(double));
    hipMalloc(&d_B, N * sizeof(double));

    hipMemcpy(d_A, host_A.data(), N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, host_B.data(), N * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((size + block.x - 1) / block.x, (size + block.y - 1) / block.y);

    int iters = 0;
    int k = 10000;
    double error = 1.0;

    hipStream_t stream;
    hipStreamCreate(&stream);   //

    hipGraph_t graph;
    hipGraphExec_t instance;

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    for (int i = 0; i < k; ++i) {
        jacobi_kernel<<<grid, block, 0, stream>>>(d_B, d_A, size);
        std::swap(d_A, d_B);
    }

    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);

    auto start = std::chrono::high_resolution_clock::now();

    while (iters < max_iters && error > eps) {
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);  //Ожидает завершения всех задач в stream, то есть ждет, пока GPU закончит вычисления, запущенные графом

        error = compute_max_error(d_A, d_B, N);
        std::swap(d_A, d_B);
        iters += k;
    }
    hipMemcpy(host_A.data(), d_A, N * sizeof(double), hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Elapsed time (ms): " << elapsed.count() * 1000 << std::endl;
    std::cout << "Iterations: " << iters << ", Error: " << error << std::endl;

    save_to_file(host_A, size, "output.txt");

    hipFree(d_A);
    hipFree(d_B);
    hipStreamDestroy(stream);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(instance);

    return 0;
}
